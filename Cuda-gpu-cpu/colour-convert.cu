#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "colour-convert.h"
#include <helper_timer.h>

//defining preprocessors for global usage
#define N (1000*700)
#define BLOCKX 1000
#define BLOCKY 700
#define WIDTH 1000
#define HEIGHT 700
#define NUM_THREADS 32

//if pixel values fall out of range re-adjust them
__device__ unsigned char clip_rgb_for_device(int x)
{
  if(x > 255)
    return 255;
  if(x < 0)
    return 0;
  return (unsigned char)x;
}

//initializing gpu
__global__ void intialize_gpu(void)
{

}

void timerKernelCall(){
  intialize_gpu<<<1,1>>>();
}

//kernel function for rgb to yuu conversion on gpu side. This takes value of
//r,g,b pixels and convert them into y,u,v values using given formula
__global__ void rgb2yuvKernel( unsigned char *d_r, unsigned char *d_g,
                               unsigned char *d_b, unsigned char *d_y,
                               unsigned char *d_u, unsigned char *d_v) {

  unsigned char r, g, b;

  //finding the index based on offset, block number and thread number
  int y = threadIdx.y+ blockIdx.y*blockDim.y;
  int x = threadIdx.x+ blockIdx.x*blockDim.x;
  int index = HEIGHT*y +x;

  if (y < WIDTH && x < HEIGHT)
  {
    r = d_r[index];
    g = d_g[index];
    b = d_b[index];

    //calculating the y,u,v from given r,g,b values
    d_y[index] = (unsigned char)( 0.299* r + 0.587* g +  0.114* b);
    d_v[index] = (unsigned char)( 0.499* r - 0.418* g - 0.0813* b + 128);
    d_u[index] = (unsigned char)(-0.169* r - 0.331* g +  0.499* b + 128);
  }
}

//kernel function for yuv to rgb conversion on gpu side. This takes value of
//y,u,v pixels and convert them into r,g,b values using given formula
__global__ void yuv2rgbKernel( unsigned char *d_y, unsigned char *d_u,
                               unsigned char *d_v, unsigned char *d_r,
                               unsigned char *d_g, unsigned char *d_b) {

  int  rt,gt,bt;
  int  yr,cb,cr;

  //finding the index based on offset, block number and thread number
  int y = threadIdx.y+ blockIdx.y*blockDim.y;
  int x = threadIdx.x+ blockIdx.x*blockDim.x;
  int index = HEIGHT*y +x;

  if (y < WIDTH && x < HEIGHT)
  {
   yr  = (int)d_y[index];
   cb = (int)d_u[index] - 128;
   cr = (int)d_v[index] - 128;

   rt  = (int)( yr + 1.402*cr);
   gt  = (int)( yr - 0.344*cb - 0.714*cr);
   bt  = (int)( yr + 1.772*cb);

   //calulating the r,g,b from above calculations
   d_r[index] = clip_rgb_for_device(rt);
   d_g[index] = clip_rgb_for_device(gt);
   d_b[index] = clip_rgb_for_device(bt);

   //uncomment lines below to test if image is white and comment 3 lines above
   // d_r[index] = 255;
   // d_g[index] = 255;
   // d_b[index] = 255;
 }
}

//universal hipMalloc API for all the future calls to hipMalloc for efficiency
void my_mallocForCuda(unsigned char **location)
{
  hipError_t error = hipSuccess;
  unsigned char * x = NULL;
  error = hipMalloc((void**) &x, sizeof(unsigned char)*N);
  if(error != hipSuccess) {
    printf("Error in allocating memory using hipMalloc %d. Exiting\n", error );
    exit(0);
  }
  *location = x;

}

//universal hipMemcpy API for all the future calls to hipMalloc for efficiency
void my_memcpyForCuda(unsigned char *o_pixel, unsigned char *i_pixel, hipMemcpyKind way)
{
  hipError_t error = hipSuccess;
  error = hipMemcpy(o_pixel, i_pixel, sizeof(unsigned char)* N , way);
  if(error != hipSuccess) {
    printf("Error in copying into memory using hipMemcpy %d. Exiting\n", error );
    exit(0);
  }
}

//universal hipFree API for all the future calls to hipMalloc for efficiency
void my_freeForCuda(unsigned char *location)
{
  hipError_t error = hipSuccess;
  error = hipFree(location);
  if(error != hipSuccess) {
    printf("Error in freeing memory using hipFree %d. Exiting\n", error );
    exit(0);
  }
}

PPM_IMG question1(PPM_IMG h_in)
{

  StopWatchInterface *timer=NULL;

  //output image storage and memory initialization
  PPM_IMG h_out;

  h_out.w = h_in.w;
  h_out.h = h_in.h;
  h_out.img_r = (unsigned char *)malloc( N * sizeof(unsigned char));
  h_out.img_g = (unsigned char *)malloc( N * sizeof(unsigned char));
  h_out.img_b = (unsigned char *)malloc( N * sizeof(unsigned char));
  if (h_out.img_r == NULL || h_out.img_g == NULL ||  h_out.img_b == NULL) {
    printf("Error in allocating memory for output image in rgb2yuvGPU. Exiting");
    exit(0);
  }

  // declare GPU memory pointers for both kinds of images
  unsigned char * d_r, * d_g, * d_b;
  unsigned char * o_r, * o_g, * o_b;

  //threads sync
  hipDeviceSynchronize();

  // allocate GPU memory for different pixels structure
  my_mallocForCuda(&d_r); my_mallocForCuda(&d_g); my_mallocForCuda(&d_b);
  my_mallocForCuda(&o_r); my_mallocForCuda(&o_g); my_mallocForCuda(&o_b);

	// transfer the array to the GPU

  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  //copying input image to GPU
  my_memcpyForCuda(d_r, h_in.img_r, hipMemcpyHostToDevice);
  my_memcpyForCuda(d_g, h_in.img_g, hipMemcpyHostToDevice);
  my_memcpyForCuda(d_b, h_in.img_b, hipMemcpyHostToDevice);

  sdkStopTimer(&timer);

  printf("Data copy time from HOST(CPU) TO DEVICE(HOST): %f (ms)\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);



  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  //now copy the output gpu yuv image to host cpu for further processing
  my_memcpyForCuda(h_out.img_r, o_r, hipMemcpyDeviceToHost);
  my_memcpyForCuda(h_out.img_g, o_g, hipMemcpyDeviceToHost);
  my_memcpyForCuda(h_out.img_b, o_b, hipMemcpyDeviceToHost);

  printf("Data copy time from DEVICE(GPU) TO HOST (CPU): %f (ms)\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);

  //free all the allocated memory
  my_freeForCuda(d_r); my_freeForCuda(d_g); my_freeForCuda(d_b);
  my_freeForCuda(o_r); my_freeForCuda(o_g); my_freeForCuda(o_b);

  hipDeviceSynchronize();
  return h_out;
}





//this funtion returns a yuv image from and input rgb ppm image
YUV_IMG rgb2yuvGPU(PPM_IMG h_in)
{

  //StopWatchInterface *timer=NULL;

  //deciding on threads and blocks for parallel processing
  const dim3 threads(NUM_THREADS,NUM_THREADS,1);
  const dim3 blocks(BLOCKX, BLOCKY, 1);

  //output image storage and memory initialization
  YUV_IMG h_out;

  h_out.w = h_in.w;
  h_out.h = h_in.h;
  h_out.img_y = (unsigned char *)malloc( N * sizeof(unsigned char));
  h_out.img_u = (unsigned char *)malloc( N * sizeof(unsigned char));
  h_out.img_v = (unsigned char *)malloc( N * sizeof(unsigned char));
  if (h_out.img_y == NULL || h_out.img_u == NULL ||  h_out.img_v == NULL) {
    printf("Error in allocating memory for output image in rgb2yuvGPU. Exiting");
    exit(0);
  }

  // declare GPU memory pointers for both kinds of images
  unsigned char * d_r, * d_g, * d_b;
  unsigned char * d_y, * d_u, * d_v;

  //threads sync
  hipDeviceSynchronize();

  // allocate GPU memory for different pixels structure
  my_mallocForCuda(&d_r); my_mallocForCuda(&d_g); my_mallocForCuda(&d_b);
  my_mallocForCuda(&d_y); my_mallocForCuda(&d_u); my_mallocForCuda(&d_v);

	// transfer the array to the GPU

  //sdkCreateTimer(&timer);
  //sdkStartTimer(&timer);

  //copying input image to GPU
  my_memcpyForCuda(d_r, h_in.img_r, hipMemcpyHostToDevice);
  my_memcpyForCuda(d_g, h_in.img_g, hipMemcpyHostToDevice);
  my_memcpyForCuda(d_b, h_in.img_b, hipMemcpyHostToDevice);

  //sdkStopTimer(&timer);

  //printf("Data copy time from HOST(CPU) TO DEVICE(HOST): %f (ms)\n", sdkGetTimerValue(&timer));
  //sdkDeleteTimer(&timer);

  // launch the kernel with given size of block and threads
  rgb2yuvKernel<<<blocks,threads>>>(d_r, d_g, d_b, d_y, d_u, d_v);

  // copy back the result array to the CPU


  //sdkCreateTimer(&timer);
  //sdkStartTimer(&timer);

  //now copy the output gpu yuv image to host cpu for further processing
  my_memcpyForCuda(h_out.img_y, d_y, hipMemcpyDeviceToHost);
  my_memcpyForCuda(h_out.img_u, d_u, hipMemcpyDeviceToHost);
  my_memcpyForCuda(h_out.img_v, d_v, hipMemcpyDeviceToHost);

  //printf("Data copy time from DEVICE(GPU) TO HOST (CPU): %f (ms)\n", sdkGetTimerValue(&timer));
  //sdkDeleteTimer(&timer);

  //free all the allocated memory
  my_freeForCuda(d_r); my_freeForCuda(d_g); my_freeForCuda(d_b);
  my_freeForCuda(d_y); my_freeForCuda(d_u); my_freeForCuda(d_v);

  hipDeviceSynchronize();

  //return yuv image
  return h_out;
}

//this funtion returns a rgb image from and input yuv image
PPM_IMG yuv2rgbGPU(YUV_IMG h_in)
{

  //StopWatchInterface *timer=NULL;

  //deciding on threads and blocks for parallel processing
  const dim3 threads(NUM_THREADS,NUM_THREADS,1);
  const dim3 blocks(BLOCKX, BLOCKY, 1);

  //output image and its memory initialization
  PPM_IMG h_out;

  h_out.w = h_in.w;
  h_out.h = h_in.h;
  h_out.img_r = (unsigned char *)malloc(sizeof(unsigned char)*N);
  h_out.img_g = (unsigned char *)malloc(sizeof(unsigned char)*N);
  h_out.img_b = (unsigned char *)malloc(sizeof(unsigned char)*N);
  if (h_out.img_r == NULL || h_out.img_g == NULL ||  h_out.img_b == NULL) {
    printf("Error in allocating memory for output image in yuv2rgbGPU. Exiting");
    exit(0);
  }

  // declare GPU memory pointers for both kinds of images
  unsigned char * d_y, * d_u, * d_v;
  unsigned char * d_r, * d_g, * d_b;

  //threads sync
  hipDeviceSynchronize();

  //allocate GPU memory for different pixels structure
  my_mallocForCuda(&d_r); my_mallocForCuda(&d_g); my_mallocForCuda(&d_b);
  my_mallocForCuda(&d_y); my_mallocForCuda(&d_u); my_mallocForCuda(&d_v);

  //sdkCreateTimer(&timer);
  //sdkStartTimer(&timer);

  //copying yuv image from host to cuda gpu
  my_memcpyForCuda(d_y, h_in.img_y, hipMemcpyHostToDevice);
  my_memcpyForCuda(d_u, h_in.img_u, hipMemcpyHostToDevice);
  my_memcpyForCuda(d_v, h_in.img_v, hipMemcpyHostToDevice);

  //sdkStopTimer(&timer);
  //printf("Data copy time from HOST(CPU) TO DEVICE(HOST): %f (ms)\n", sdkGetTimerValue(&timer));
  //sdkDeleteTimer(&timer);

  //calling kernel function with blocks and threads
  yuv2rgbKernel<<<blocks,threads>>>(d_y, d_u, d_v, d_r, d_g, d_b);

  //sdkCreateTimer(&timer);
  //sdkStartTimer(&timer);

  //now copy the output gpu rgb image to host cpu for further processing
  my_memcpyForCuda(h_out.img_r, d_r, hipMemcpyDeviceToHost);
  my_memcpyForCuda(h_out.img_g, d_g, hipMemcpyDeviceToHost);
  my_memcpyForCuda(h_out.img_b, d_b, hipMemcpyDeviceToHost);

  //printf("Data copy time from DEVICE(GPU) TO HOST (CPU): %f (ms)\n", sdkGetTimerValue(&timer));
  //sdkDeleteTimer(&timer);

  //freeing all allocated memory
  my_freeForCuda(d_r); my_freeForCuda(d_g); my_freeForCuda(d_b);
  my_freeForCuda(d_y); my_freeForCuda(d_u); my_freeForCuda(d_v);

  //returning final image
  return h_out;
}

//Convert RGB to YUV444, all components in [0, 255]
YUV_IMG rgb2yuv(PPM_IMG img_in)
{
    YUV_IMG img_out;
    int i;//, j;
    unsigned char r, g, b;
    unsigned char y, cb, cr;

    img_out.w = img_in.w;
    img_out.h = img_in.h;
    img_out.img_y = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_u = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_v = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);

    for(i = 0; i < img_out.w*img_out.h; i ++){
        r = img_in.img_r[i];
        g = img_in.img_g[i];
        b = img_in.img_b[i];

        y  = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
        cb = (unsigned char)(-0.169*r - 0.331*g +  0.499*b + 128);
        cr = (unsigned char)( 0.499*r - 0.418*g - 0.0813*b + 128);

        img_out.img_y[i] = y;
        img_out.img_u[i] = cb;
        img_out.img_v[i] = cr;
    }

    return img_out;
}

unsigned char clip_rgb(int x)
{
    if(x > 255)
        return 255;
    if(x < 0)
        return 0;

    return (unsigned char)x;
}

//Convert YUV to RGB, all components in [0, 255]
PPM_IMG yuv2rgb(YUV_IMG img_in)
{
    PPM_IMG img_out;
    int i;
    int  rt,gt,bt;
    int y, cb, cr;


    img_out.w = img_in.w;
    img_out.h = img_in.h;
    img_out.img_r = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_g = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);
    img_out.img_b = (unsigned char *)malloc(sizeof(unsigned char)*img_out.w*img_out.h);

    for(i = 0; i < img_out.w*img_out.h; i ++){
        y  = (int)img_in.img_y[i];
        cb = (int)img_in.img_u[i] - 128;
        cr = (int)img_in.img_v[i] - 128;

        rt  = (int)( y + 1.402*cr);
        gt  = (int)( y - 0.344*cb - 0.714*cr);
        bt  = (int)( y + 1.772*cb);

        img_out.img_r[i] = clip_rgb(rt);
        img_out.img_g[i] = clip_rgb(gt);
        img_out.img_b[i] = clip_rgb(bt);
    }

    return img_out;
}
